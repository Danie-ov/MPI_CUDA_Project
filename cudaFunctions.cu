#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "myProto.h"
#include <stdio.h>
#include <string.h>



__device__ int checkStrChr(char* str, char ch)
{
    while(*str != NULL)
    {
        if(*str == ch)
            return 1;
        
        str++;
    }
    return 0;
}

__device__ int checkFirstGroupGPU(char a, char b)
{
   
   char* firstGroup[FIRST_GROUP_SIZE] = { "NDEQ", "NEQK", "STA", "MILV", "QHRK", "NHQK", "FYW", "HY", "MILF" };
   int i;
   for(i=0; i < FIRST_GROUP_SIZE; i++)
   {
      if(checkStrChr(firstGroup[i], a) && checkStrChr(firstGroup[i], b))
            return 1;
   }
   return 0;
}

__device__ int checkSecondGroupGPU(char a, char b)
{

    char* secondGroup[SECOND_GROUP_SIZE] = { "SAG", "ATV", "CSA", "SGND", "STPA", "STNK", "NEQHRK", "NDEQHK", "SNDEQK", "HFY", "FVLIM" };
   int i;
   for(i=0; i < SECOND_GROUP_SIZE; i++)
   {
      if(checkStrChr(secondGroup[i], a) && checkStrChr(secondGroup[i], b))
            return 1;
   }
   return 0;
}


__global__  void calcScore(int* weight, char* firstSeq, char* sequence, int firstSeqLen, int secondSeqLen, int* scores, int* offsets, int* mutants) {
    
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    int i, score = 0, offset = id / secondSeqLen, mutant = id % secondSeqLen + 1;
    char a, b;
    
    for(i=0; i <= secondSeqLen; i++)
    {
        a = firstSeq[i+offset];
        if(i < mutant)
            b = sequence[i];
            
        else if(i > mutant)
            b = sequence[i-1];
            
        if(i != mutant)
        {
            if(a == b)
                score += weight[0];
            
            else if(checkFirstGroupGPU(a, b))
            {
                score -= weight[1];

            }
            
            else if(checkSecondGroupGPU(a, b))
                score -= weight[2];
            else
                score -= weight[3];
        }    
    }
    scores[id] = score;
    offsets[id] = offset;
    mutants[id] = mutant;
}

int computeOnGPU(int* weight, char* firstSeq, char** sequences, int numOfSeqs, int** maxScore, int** maxOffset, int** maxMutant, int* sizeMat) {
    
    int* dev_weight = 0, *dev_max_score = 0, *dev_max_offset = 0, *dev_max_mutant = 0;
    char* dev_firstSeq = 0;
    int firstSeqLen = strlen(firstSeq), i;
    char* dev_sequence = 0;
    int seqLen = strlen(sequences[0]);
    hipError_t cuda_status;
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    int numThreadsPerBlock, numBlocks, extraBlock;

    cuda_status = hipMalloc((void**)&dev_weight, sizeof(int)*4);
    if(cuda_status != hipSuccess)
    {
        printf("CUDA STATUS1 ERROR!\n");
        hipFree(dev_max_score);
        hipFree(dev_max_offset);
        hipFree(dev_max_mutant);
        hipFree(dev_weight);
        return EXIT_FAILURE;
    }

    cuda_status = hipMemcpy(dev_weight, weight, 4*sizeof(int), hipMemcpyHostToDevice);
    if(cuda_status != hipSuccess)
    {
        printf("CUDA STATUS2 ERROR!\n");
        hipFree(dev_max_score);
        hipFree(dev_max_offset);
        hipFree(dev_max_mutant);
        hipFree(dev_weight);
        return EXIT_FAILURE;
    }

    cuda_status = hipMalloc((void**)&dev_firstSeq, sizeof(char)*firstSeqLen);
    if(cuda_status != hipSuccess)
    {
        printf("CUDA STATUS3 ERROR\n");
        hipFree(dev_max_score);
        hipFree(dev_max_offset);
        hipFree(dev_max_mutant);
        hipFree(dev_weight);
        return EXIT_FAILURE;
    }

    cuda_status = hipMemcpy(dev_firstSeq, firstSeq, firstSeqLen, hipMemcpyHostToDevice);
    if(cuda_status != hipSuccess)
    {
        printf("CUDA STATUS4 ERROR!\n");
        hipFree(dev_max_score);
        hipFree(dev_max_offset);
        hipFree(dev_max_mutant);
        hipFree(dev_weight);
        hipFree(dev_firstSeq);
        return EXIT_FAILURE;
    }

    for(i=0; i < numOfSeqs; i++)
    {
        int secondSeqLen = strlen(sequences[i]);
        int size = (firstSeqLen - secondSeqLen) * secondSeqLen;

        numThreadsPerBlock = prop.maxThreadsPerBlock < size ? prop.maxThreadsPerBlock : size;
        numBlocks = size / numThreadsPerBlock;
        extraBlock = size % numThreadsPerBlock != 0;

        cuda_status = hipMalloc((void**)&dev_max_score, sizeof(int)*size);
        if(cuda_status != hipSuccess)
        {
            printf("CUDA SCORE ERROR!\n");
            return EXIT_FAILURE;
        }
        cuda_status = hipMalloc((void**)&dev_max_offset, sizeof(int)*size);
        if(cuda_status != hipSuccess)
        {
            printf("CUDA OFFSET ERROR!\n");
            hipFree(dev_max_score);
            return EXIT_FAILURE;
        }
        cuda_status = hipMalloc((void**)&dev_max_mutant, sizeof(int)*size);
        if(cuda_status != hipSuccess)
        {
            printf("CUDA MUTANT ERROR!\n");
            hipFree(dev_max_score);
            hipFree(dev_max_offset);
            return EXIT_FAILURE;
        }

        cuda_status = hipMalloc((void**)&dev_sequence, sizeof(char)*secondSeqLen);
        if(cuda_status != hipSuccess)
        {
            printf("CUDA STATUS6 ERROR!\n");
            hipFree(dev_weight);
            hipFree(dev_firstSeq);
            return EXIT_FAILURE;
        }

        cuda_status = hipMemcpy(dev_sequence, sequences[i], sizeof(char)*secondSeqLen, hipMemcpyHostToDevice);
        if(cuda_status != hipSuccess)
        {
            printf("CUDA STATUS7 ERROR!\n");
            hipFree(dev_weight);
            hipFree(dev_firstSeq);
            return EXIT_FAILURE;
        }

        calcScore<<<numBlocks+extraBlock, numThreadsPerBlock>>>(dev_weight, dev_firstSeq, dev_sequence, firstSeqLen, seqLen, dev_max_score, dev_max_offset, dev_max_mutant);

        cuda_status = hipDeviceSynchronize();
        if(cuda_status != hipSuccess)
        {
            printf("CUDA SYNCRONIZE ERROR!\n");
            hipFree(dev_weight);
            hipFree(dev_firstSeq);
            return EXIT_FAILURE;
        }

        maxScore[i] = (int*)malloc(sizeof(int)*size);
        if(!maxScore[i])
            return EXIT_FAILURE;

        cuda_status = hipMemcpy(maxScore[i], dev_max_score, sizeof(int)*size, hipMemcpyDeviceToHost);
        if(cuda_status != hipSuccess)
        {
            printf("CUDA SCORE1 ERROR!\n");
            hipFree(dev_weight);
            hipFree(dev_firstSeq);
            return EXIT_FAILURE;
        }

        maxOffset[i] = (int*)malloc(sizeof(int)*size);
        if(!maxOffset[i])
            return EXIT_FAILURE;
        
        cuda_status = hipMemcpy(maxOffset[i], dev_max_offset, sizeof(int)*size, hipMemcpyDeviceToHost);
        if(cuda_status != hipSuccess)
        {
            printf("CUDA OFFSET1 ERROR!\n");
            hipFree(dev_weight);
            hipFree(dev_firstSeq);
            return EXIT_FAILURE;
        }

        maxMutant[i] = (int*)malloc(sizeof(int)*size);
        if(!maxMutant[i])
            return EXIT_FAILURE;

        cuda_status = hipMemcpy(maxMutant[i], dev_max_mutant, sizeof(int)*size, hipMemcpyDeviceToHost);
        if(cuda_status != hipSuccess)
        {
            printf("CUDA MUTANT1 ERROR!\n");
            hipFree(dev_weight);
            hipFree(dev_firstSeq);
            return EXIT_FAILURE;
        }

        sizeMat[i] = size;

    }

    hipFree(dev_max_score);
    hipFree(dev_max_offset);
    hipFree(dev_max_mutant);
    hipFree(dev_firstSeq);
    hipFree(dev_weight);
    hipFree(sequences);

    return EXIT_SUCCESS;
}

